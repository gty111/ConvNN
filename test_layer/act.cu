#include "Layer.h"

int main(){

    Tensor<float> in(1,1,3,3);
    in.initData(0,1);
    in.print();

    Activation *conv = new Activation(HIPDNN_ACTIVATION_RELU,&in);
    conv->initData(0,1);
    conv->forward();
    hipDeviceSynchronize();
    conv->_out->print();
    

    conv->_out->initGrad(0,1);
    conv->backward();
    hipDeviceSynchronize();
    conv->_in->printgrad();
}