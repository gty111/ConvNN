// use 1x1 conv to replace full connected
// https://forums.developer.nvidia.com/t/fully-connected-layer-using-cudnn-library/66998

#include <hip/hip_runtime.h>
#include <cudnn.h>
#include <random>

std::random_device rd;  //Will be used to obtain a seed for the random number engine
std::mt19937 gen(rd()); //Standard mersenne_twister_engine seeded with rd()
std::normal_distribution<float> dis(0,1);

int main(){
    // create handle 
    cudnnHandle_t cuhandle;
    cudnnCreate(&cuhandle);

    // create tensor descriptor
    int n=1,c=1,h=5,w=5,out_size=10;
    cudnnTensorDescriptor_t tensor_des;
    cudnnCreateTensorDescriptor(&tensor_des);
    cudnnSetTensor4dDescriptor(tensor_des, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, n, c, h, w);

    // create out_tensor descriptor
    cudnnTensorDescriptor_t out_tensor_des;
    cudnnCreateTensorDescriptor(&out_tensor_des);
    cudnnSetTensor4dDescriptor(out_tensor_des, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, 1, out_size, 1, 1);

    // create conv function descriptor
    cudnnConvolutionDescriptor_t conv_des;
    cudnnCreateConvolutionDescriptor(&conv_des);
    cudnnSetConvolution2dDescriptor(conv_des,0,0,1,1,1,1,CUDNN_CONVOLUTION,CUDNN_DATA_FLOAT);

    // create filter descripter
    int filter_w=5,filter_h=5;
    cudnnFilterDescriptor_t filter_des;
    cudnnCreateFilterDescriptor(&filter_des);
    cudnnSetFilter4dDescriptor(filter_des,CUDNN_DATA_FLOAT,CUDNN_TENSOR_NCHW,out_size,c,filter_h,filter_w);

    // create input
    float* tensor;
    hipMallocManaged(&tensor,n*c*h*w*sizeof(float));
    for(int i=0;i<h;i++){
        for(int j=0;j<w;j++){
            tensor[i*w+j] = 1;
            printf("%f ",tensor[i*w+j]);
        }
        printf("\n");
    }

    // create output
    float* out_tensor;
    hipMallocManaged(&out_tensor,n*out_size*sizeof(float));

    // create filter
    float *tensor_filter;
    hipMallocManaged(&tensor_filter,c*out_size*filter_w*filter_h*sizeof(float));
    for(int i=0;i<c*out_size*filter_w*filter_h;i++){
        tensor_filter[i] = 1;
    }
    
    float alpha=1,beta=0;
    cudnnConvolutionForward( cuhandle
                            ,&alpha
                            ,tensor_des
                            ,tensor
                            ,filter_des
                            ,tensor_filter
                            ,conv_des
                            ,CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM
                            ,nullptr
                            ,0
                            ,&beta
                            ,out_tensor_des
                            ,out_tensor
                            );

    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();
    if(err!=hipSuccess){
        printf("%d\n",err);
    }
    
    for(int i=0;i<out_size;i++){
        printf("%f ",out_tensor[i]);
    }
    printf("\n");
}